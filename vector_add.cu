
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define VECTOR_LENGTH 10000
#define MAX_ERR 1e-4

__global__ void vector_add(float *out, float *a, float *b, int n){
    for(int i=0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // step1: allocate memory on CPU
    a = (float*)malloc(sizeof(float)*VECTOR_LENGTH);
    b = (float*)malloc(sizeof(float)*VECTOR_LENGTH);
    out = (float*)malloc(sizeof(float)*VECTOR_LENGTH);

    // step2: data initilization
    for(int i = 0; i < VECTOR_LENGTH; i++){
        a[i] = 3.0f;
        b[i] = 0.14f;
    }

    // step3: allocate memory on GPU
    hipMalloc((void**)&d_a, sizeof(float)*VECTOR_LENGTH);
    hipMalloc((void**)&d_b, sizeof(float)*VECTOR_LENGTH);
    hipMalloc((void**)&d_out, sizeof(float)*VECTOR_LENGTH);

    // step4: transfer input data from host(CPU) to device(GPU) memory
    hipMemcpy(d_a, a, sizeof(float)*VECTOR_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*VECTOR_LENGTH, hipMemcpyHostToDevice);

    // step5: execute kernel function on GPU
    vector_add<<<1, 1>>>(d_out, d_a, d_b, VECTOR_LENGTH);

    // step6: transfer output from device(GPU) memory to host(CPU)
    hipMemcpy(out, d_out, sizeof(float)*VECTOR_LENGTH, hipMemcpyDeviceToHost);

    for(int i = 0; i < VECTOR_LENGTH; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("out[0] is %f\n", out[0]);
    printf("PASSED\n");

    // step7: free the memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
}